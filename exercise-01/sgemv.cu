#include "hipblas.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>

int main() {
  constexpr size_t ROWS = 6;
  constexpr size_t COLUMNS = 5;
  constexpr float ALPHA = 1.0f;
  constexpr float BETA = 0.0f;

  hipblasHandle_t handle;

  std::vector<float> hostA(ROWS * COLUMNS);
  std::vector<float> hostB(COLUMNS);
  std::vector<float> hostC(ROWS);

  int index = 11;
  for (size_t i = 0; i < COLUMNS; i++) {
    for (size_t j = 0; j < ROWS; j++) {
      hostA[(i * ROWS) + j] = static_cast<float>(index++);
    }
  }

  std::fill(std::begin(hostB), std::end(hostB), 1.0f);

  // hostA:
  // [11, 17, 23, 29, 35]
  // [12, 18, 24, 30, 36]
  // [13, 19, 25, 31, 37]
  // [14, 20, 26, 32, 38]
  // [15, 21, 27, 33, 39]
  // [16, 22, 28, 34, 40]

  // hostB:
  // [1, 1, 1, 1, 1]

  // hostC:
  // [0, 0, 0, 0, 0, 0]

  float *deviceA = nullptr;
  float *deviceB = nullptr;
  float *deviceC = nullptr;

  hipMalloc((void **)&deviceA, ROWS * COLUMNS * sizeof(float));
  hipMalloc((void **)&deviceB, COLUMNS * sizeof(float));
  hipMalloc((void **)&deviceC, ROWS * sizeof(float));

  hipblasCreate(&handle);

  hipblasSetMatrix(ROWS, COLUMNS, sizeof(float), hostA.data(), ROWS, deviceA,
                  ROWS);
  hipblasSetVector(COLUMNS, sizeof(float), hostB.data(), 1, deviceB, 1);
  hipblasSetVector(ROWS, sizeof(float), hostC.data(), 1, deviceC, 1);
  hipblasSgemv(handle, HIPBLAS_OP_N, ROWS, COLUMNS, &ALPHA, deviceA, ROWS,
              deviceB, 1, &BETA, deviceC, 1);
  hipblasGetVector(ROWS, sizeof(float), deviceC, 1, hostC.data(), 1);

  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  assert(hostC[0] == 115); // [11, 17, 23, 29, 35]     [1]
  assert(hostC[1] == 120); // [12, 18, 24, 30, 36]     [1]
  assert(hostC[2] == 125); // [13, 19, 25, 31, 37]  *  [1]
  assert(hostC[3] == 130); // [14, 20, 26, 32, 38]     [1]
  assert(hostC[4] == 135); // [15, 21, 27, 33, 39]     [1]
  assert(hostC[5] == 140); // [16, 22, 28, 34, 40]

  hipblasDestroy(handle);
}